// Using CUDA device to calculate pi
#include <stdio.h>
#include <hip/hip_runtime.h>
extern "C" double getTime(void);

#define NBIN 1000000000  // Number of bins
//#define num_block  (2*56)  // Number of thread blocks
//#define num_thread  (2*8)  // Number of threads per block

// Kernel that executes on the CUDA device
__global__ void cal_pi(double *sum, int nbin, double step, int nthreads, int nblocks) {
	int i;
	double x;
	int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
	for (i=idx; i< nbin; i+=nthreads*nblocks) {
		x = (i+0.5)*step;
		sum[idx] += 4.0/(1.0+x*x);
	}
}

// Main routine that executes on the host
int main(int argc, char **argv) {

	int c;
	int num_block;
	int num_thread;
	//printf("initial argc = %d\n", argc);
	while(argc--){
		c = argc;

		//printf("c = %d\n", argc);
		printf("%d\n", atoi(*argv++));
		if (c == 2) num_block = atoi(*argv); 
		if (c == 1) num_thread = atoi(*argv);
	}

	dim3 dimGrid(num_block,1,1);  // Grid dimensions
	dim3 dimBlock(num_thread,1,1);  // Block dimensions
	double *sumHost, *sumDev;  // Pointer to host & device arrays
	double pi = 0;
	int tid;

	double step = 1.0/NBIN;  // Step size
	size_t size = num_block*num_thread*sizeof(double);  //Array memory size
	sumHost = (double *)malloc(size);  //  Allocate array on host
	hipMalloc((void **) &sumDev, size);  // Allocate array on device
   	double start = getTime();

	//printf("INITIAL PRINT 1= %s and 2 = %s\n", argv,argv-1 );

	// Initialize array in device to 0
	hipMemset(sumDev, 0, size);
	// Do calculation on device
	cal_pi <<<dimGrid, dimBlock>>> (sumDev, NBIN, step, num_thread, num_block); // call CUDA kernel
	// Retrieve result from device and store it in host array
	hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
	for(tid=0; tid<num_thread*num_block; tid++)
		pi += sumHost[tid];
	pi *= step;

	// Print results
	double delta = getTime() - start;
	printf("PI = %.16g computed in %.4g seconds with NUM_BLOCK = %d and NUM_THREAD = %d\n", pi, delta, num_block, num_thread);
	// Cleanup
	free(sumHost);
	hipFree(sumDev);

	return 0;
}

